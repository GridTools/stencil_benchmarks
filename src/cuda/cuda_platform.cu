#include "cuda/cuda_platform.h"

#include "cuda/cuda_hdiff_variant.h"
#include "cuda/cuda_hdiff_variant_ijnoshared.h"
#include "cuda/cuda_hdiff_variant_incache.h"
#include "cuda/cuda_hdiff_variant_noshared.h"
#include "cuda/cuda_vadv_variant.h"
#include "cuda/cuda_variant_1d.h"
#include "cuda/cuda_variant_ij_blocked.h"
#include "cuda/cuda_variant_ijk_blocked.h"

namespace platform {

    namespace cuda {

        void cuda::setup(arguments &args) {
            auto &basic = args.command("basic", "variant");
            basic.command("1d").add("blocksize", "1D block size", "32");
            basic.command("ij-blocked")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8");
            basic.command("ijk-blocked")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8")
                .add("k-blocksize", "block size in k-direction", "8");
            auto &hdiff = args.command("hdiff", "variant");
            hdiff.command("ij-blocked")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8");
            hdiff.command("ij-blocked-noshared")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8");
            hdiff.command("ijk-blocked-noshared")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8")
                .add("k-blocksize", "block size in k-direction", "8");
            hdiff.command("ijk-blocked-incache")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8")
                .add("k-blocksize", "block size in k-direction", "1");
            auto &vadv = args.command("vadv", "variant");
            vadv.command("ij-blocked")
                .add("i-blocksize", "block size in i-direction", "32")
                .add("j-blocksize", "block size in j-direction", "8");
        }

        namespace {
            template <class ValueType>
            variant_base *create_variant_by_prec(const arguments_map &args) {
                std::string grp = args.get("group");
                std::string var = args.get("variant");

                if (grp == "basic") {
                    if (var == "1d")
                        return new variant_1d<ValueType>(args);
                    if (var == "ij-blocked")
                        return new variant_ij_blocked<ValueType>(args);
                    if (var == "ijk-blocked")
                        return new variant_ijk_blocked<ValueType>(args);
                }
                if (grp == "hdiff") {
                    if (var == "ij-blocked")
                        return new hdiff_variant<ValueType>(args);
                    if (var == "ij-blocked-noshared")
                        return new hdiff_variant_ijnoshared<ValueType>(args);
                    if (var == "ijk-blocked-noshared")
                        return new hdiff_variant_noshared<ValueType>(args);
                    if (var == "ijk-blocked-incache")
                        return new hdiff_variant_incache<ValueType>(args);
                }
                if (grp == "vadv") {
                    if (var == "ij-blocked")
                        return new vadv_variant<ValueType>(args);
                }
                return nullptr;
            }
        }

        variant_base *cuda::create_variant(const arguments_map &args) {
            std::string prec = args.get("precision");

            if (prec == "single") {
                return create_variant_by_prec<float>(args);
            } else if (prec == "double") {
                return create_variant_by_prec<double>(args);
            }

            return nullptr;
        }

        void cuda::limit_blocksize(int &iblocksize, int &jblocksize) {
            int kblocksize = 1;
            limit_blocksize(iblocksize, jblocksize, kblocksize);
        }

        void cuda::limit_blocksize(int &iblocksize, int &jblocksize, int &kblocksize) {
            if (iblocksize <= 0 || jblocksize <= 0 || kblocksize <= 0)
                throw ERROR("invalid CUDA block size");

            hipError_t err;
            int device;
            if ((err = hipGetDevice(&device)) != hipSuccess)
                throw ERROR("error in hipGetDevice: " + std::string(hipGetErrorString(err)));
            hipDeviceProp_t prop;
            if ((err = hipGetDeviceProperties(&prop, device)) != hipSuccess)
                throw ERROR("error in hipGetDeviceProperties: " + std::string(hipGetErrorString(err)));

            int iblocksize0 = iblocksize, jblocksize0 = jblocksize, kblocksize0 = kblocksize;
            bool adapt = false;
            if (iblocksize > prop.maxThreadsDim[0]) {
                iblocksize = prop.maxThreadsDim[0];
                adapt = true;
            }
            if (jblocksize > prop.maxThreadsDim[1]) {
                jblocksize = prop.maxThreadsDim[1];
                adapt = true;
            }
            if (kblocksize > prop.maxThreadsDim[2]) {
                kblocksize = prop.maxThreadsDim[2];
                adapt = true;
            }

            while (iblocksize * jblocksize * kblocksize > prop.maxThreadsPerBlock) {
                if (iblocksize > jblocksize) {
                    if (iblocksize > kblocksize)
                        iblocksize /= 2;
                    else
                        kblocksize /= 2;
                } else {
                    if (jblocksize > kblocksize)
                        jblocksize /= 2;
                    else
                        kblocksize /= 2;
                }
                adapt = true;
            }
            if (adapt) {
                std::cerr << "WARNING: adapted CUDA block size to conform to device limits "
                          << "(" << iblocksize0 << "x" << jblocksize0 << "x" << kblocksize0 << " to " << iblocksize
                          << "x" << jblocksize << "x" << kblocksize << ")" << std::endl;
            }

            if (iblocksize <= 0 || jblocksize <= 0 || kblocksize <= 0)
                throw ERROR("CUDA block size adaption failed");
        }

    } // namespace cuda

} // namespace platform
